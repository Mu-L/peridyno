#include "hip/hip_runtime.h"
#include "DiscreteElementsToTriangleSet.h"

namespace dyno
{
	typedef typename ::dyno::TOrientedBox3D<Real> Box3D;

	template<typename TDataType>
	DiscreteElementsToTriangleSet<TDataType>::DiscreteElementsToTriangleSet()
		: TopologyMapping()
	{
		mStandardSphere.loadObjFile(getAssetPath() + "standard/standard_icosahedron.obj");
		mStandardCapsule.loadObjFile(getAssetPath() + "standard/standard_capsule.obj");
	}

	template<typename Triangle>
	__global__ void SetupCubeInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Box3D> boxes,
		uint pointOffset,
		uint indexOffset,
		uint cubeOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boxes.size()) return;
		
		int idx = tId;
		Box3D box = boxes[idx];

		Vec3f hx = box.u * box.extent[0];
		Vec3f hy = box.v * box.extent[1];
		Vec3f hz = box.w * box.extent[2];

		Vec3f hyz = hy + hz;
		Vec3f hxy = hx + hy;
		Vec3f hxz = hx + hz;

		Vec3f c = box.center;

		Vec3f v0 = c - hx - hyz;
		Vec3f v1 = c + hx - hyz;
		Vec3f v2 = c + hxz - hy;
		Vec3f v3 = c - hxy + hz;

		Vec3f v4 = c - hxz + hy;
		Vec3f v5 = c + hxy - hz;
		Vec3f v6 = c + hx + hyz;
		Vec3f v7 = c - hx + hyz;

		vertices[pointOffset + idx * 8] = v0;
		vertices[pointOffset + idx * 8 + 1] = v1;
		vertices[pointOffset + idx * 8 + 2] = v2;
		vertices[pointOffset + idx * 8 + 3] = v3;
		vertices[pointOffset + idx * 8 + 4] = v4;
		vertices[pointOffset + idx * 8 + 5] = v5;
		vertices[pointOffset + idx * 8 + 6] = v6;
		vertices[pointOffset + idx * 8 + 7] = v7;

		uint offset = idx * 8 + pointOffset;

		indices[indexOffset + idx * 12] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 12 + 1] = Triangle(offset + 0, offset + 2, offset + 3);

		indices[indexOffset + idx * 12 + 2] = Triangle(offset + 0, offset + 4, offset + 5);
		indices[indexOffset + idx * 12 + 3] = Triangle(offset + 0, offset + 5, offset + 1);

		indices[indexOffset + idx * 12 + 4] = Triangle(offset + 4, offset + 7, offset + 6);
		indices[indexOffset + idx * 12 + 5] = Triangle(offset + 4, offset + 6, offset + 5);

		indices[indexOffset + idx * 12 + 6] = Triangle(offset + 1, offset + 5, offset + 6);
		indices[indexOffset + idx * 12 + 7] = Triangle(offset + 1, offset + 6, offset + 2);

		indices[indexOffset + idx * 12 + 8] = Triangle(offset + 2, offset + 6, offset + 7);
		indices[indexOffset + idx * 12 + 9] = Triangle(offset + 2, offset + 7, offset + 3);

		indices[indexOffset + idx * 12 + 10] = Triangle(offset + 0, offset + 3, offset + 7);
		indices[indexOffset + idx * 12 + 11] = Triangle(offset + 0, offset + 7, offset + 4);
	}

	template<typename Triangle>
	__global__ void SetupTetInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Tet3D> tets,
		uint pointOffset,
		uint indexOffset,
		uint tetOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;
	
		int idx = tId;
		Tet3D tet = tets[idx];

		Vec3f v0 = tet.v[0];
		Vec3f v1 = tet.v[1];
		Vec3f v2 = tet.v[2];
		Vec3f v3 = tet.v[3];

		vertices[pointOffset + idx * 4] = v0;
		vertices[pointOffset + idx * 4 + 1] = v1;
		vertices[pointOffset + idx * 4 + 2] = v2;
		vertices[pointOffset + idx * 4 + 3] = v3;

		uint offset = idx * 4 + pointOffset;

		indices[indexOffset + idx * 4] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 4 + 1] = Triangle(offset + 0, offset + 1, offset + 3);
		indices[indexOffset + idx * 4 + 2] = Triangle(offset + 1, offset + 2, offset + 3);
		indices[indexOffset + idx * 4 + 3] = Triangle(offset + 0, offset + 2, offset + 3);
	}

	__global__ void SetupVerticesForSphereInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> sphereVertices,
		DArray<Sphere3D> sphereInstances,
		uint pointOffset,
		uint sphereOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereVertices.size()) return;

		uint instanceId = tId / sphereVertices.size();
		uint vertexId = tId % sphereVertices.size();

		Sphere3D sphere = sphereInstances[instanceId];

		Vec3f v = sphereVertices[vertexId];
		vertices[pointOffset + tId] = sphere.center + sphere.radius * sphere.rotation.rotate(v);
	}

	template<typename Triangle>
	__global__ void SetupIndicesForSphereInstances(
		DArray<Triangle> indices,
		DArray<Triangle> sphereIndices,
		DArray<Sphere3D> sphereInstances,
		uint vertexSize,						//vertex size of the instance sphere 
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereIndices.size()) return;

		uint instanceId = tId / sphereIndices.size();
		uint indexId = tId % sphereIndices.size();

		int vertexOffset = indexOffset + instanceId * vertexSize;
		
		Triangle tIndex = sphereIndices[indexId];
		indices[indexOffset + tId] = Triangle(tIndex[0] + vertexOffset, tIndex[1] + vertexOffset, tIndex[2] + vertexOffset);
	}

	__global__ void SetupVerticesForCapsuleInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> capsuleVertices,
		DArray<Capsule3D> capsuleInstances,
		uint pointOffset,
		uint capsuleOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size() * capsuleVertices.size()) return;

		uint instanceId = tId / capsuleVertices.size();
		uint vertexId = tId % capsuleVertices.size();

		Capsule3D capsule = capsuleInstances[instanceId];
		float r = capsule.radius;
		float h = capsule.halfLength;
		auto rot = capsule.rotation.toMatrix3x3();
		Vec3f center = capsule.center; 

		Vec3f v = capsuleVertices[vertexId];
		Vec3f orignZ = Vec3f(0, 1, 0);
		Vec3f newZ = Vec3f(0, h, 0);

		if (v.y >= 1)
		{
			vertices[pointOffset + tId] = rot * ((v - orignZ) * r + newZ) + center;
		}
		else if (v.y <= -1) 
		{
			vertices[pointOffset + tId] = rot * ((v + orignZ) * r - newZ) + center;
		}
		else
		{
			vertices[pointOffset + tId] = rot * (v * Vec3f(r, h, r)) + center;
		}	
	}

	template<typename Triangle>
	__global__ void SetupIndicesForCapsuleInstances(
		DArray<Triangle> indices,
		DArray<Triangle> capsuleIndices,
		DArray<Capsule3D> capsuleInstances,
		uint vertexSize,						//vertex size of the instance sphere 
		uint vertexOffset,
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size() * capsuleIndices.size()) return;

		uint instanceId = tId / capsuleIndices.size();
		uint indexId = tId % capsuleIndices.size();

		vertexOffset += instanceId * vertexSize;
		
		Triangle tIndex = capsuleIndices[indexId];
		indices[indexOffset + tId] = Triangle(tIndex[0] + vertexOffset, tIndex[1] + vertexOffset, tIndex[2] + vertexOffset);
	}

	template<typename TDataType>
	bool DiscreteElementsToTriangleSet<TDataType>::apply()
	{
		if (this->outTriangleSet()->isEmpty())
		{
			this->outTriangleSet()->allocate();
		}

		auto inTopo = this->inDiscreteElements()->constDataPtr();

		DArray<Box3D>& boxInGlobal = inTopo->boxesInGlobal();
		DArray<Sphere3D>& sphereInGlobal = inTopo->spheresInGlobal();
		DArray<Tet3D>& tetInGlobal = inTopo->tetsInGlobal();
		DArray<Capsule3D>& capsuleInGlobal = inTopo->capsulesInGlobal();

		ElementOffset elementOffset = inTopo->calculateElementOffset();

		int numOfSpheres = sphereInGlobal.size();
		int numofCaps = capsuleInGlobal.size();
		int numOfBoxes = boxInGlobal.size();
		int numOfTets = tetInGlobal.size();
		
		auto triSet = this->outTriangleSet()->getDataPtr();

		auto& vertices = triSet->getPoints();
		auto& indices = triSet->triangleIndices();

		auto& sphereVertices = mStandardSphere.getPoints();
		auto& sphereIndices = mStandardSphere.triangleIndices();

		auto& capsuleVertices = mStandardCapsule.getPoints();
		auto& capsuleIndices = mStandardCapsule.triangleIndices();
		
		int numOfVertices = 8 * numOfBoxes + 4 * numOfTets + sphereVertices.size() * numOfSpheres + capsuleVertices.size() * numofCaps;
		int numOfTriangles = 12 * numOfBoxes + 4 * numOfTets + sphereIndices.size() * numOfSpheres + capsuleIndices.size() * numofCaps;

		vertices.resize(numOfVertices);
		indices.resize(numOfTriangles);

		uint vertexOffset = 0;
		uint indexOffset = 0;

		//Setup spheres
		cuExecute(numOfSpheres * sphereVertices.size(),
			SetupVerticesForSphereInstances,
			vertices,
			sphereVertices,
			sphereInGlobal,
			vertexOffset,
			elementOffset.sphereIndex());

		cuExecute(numOfSpheres * sphereIndices.size(),
			SetupIndicesForSphereInstances,
			indices,
			sphereIndices,
			sphereInGlobal,
			sphereVertices.size(),
			indexOffset);

		vertexOffset += numOfSpheres * sphereVertices.size();
		indexOffset += numOfSpheres * sphereIndices.size();

		//Setup boxes
		cuExecute(numOfBoxes,
			SetupCubeInstances,
			vertices,
			indices,
			boxInGlobal,
			vertexOffset,
			indexOffset,
			elementOffset.boxIndex());

		vertexOffset += numOfBoxes * 8;
		indexOffset += numOfBoxes * 12;

		//Setup tets
		cuExecute(numOfTets,
			SetupTetInstances,
			vertices,
			indices,
			tetInGlobal,
			vertexOffset,
			indexOffset,
			elementOffset.tetIndex());

		vertexOffset += numOfTets * 4;
		indexOffset += numOfTets * 4;

		cuExecute(numofCaps * capsuleVertices.size(),
			SetupVerticesForCapsuleInstances,
			vertices,
			capsuleVertices,
			capsuleInGlobal,
			vertexOffset,
			elementOffset.capsuleIndex());

		cuExecute(numofCaps * capsuleIndices.size(),
			SetupIndicesForCapsuleInstances,
			indices,
			capsuleIndices,
			capsuleInGlobal,
			capsuleVertices.size(),
			vertexOffset,
			indexOffset);

		vertexOffset += numofCaps * capsuleVertices.size();
		indexOffset += numofCaps * capsuleIndices.size();

		this->outTriangleSet()->getDataPtr()->update();

		return true;
	}

	DEFINE_CLASS(DiscreteElementsToTriangleSet);
}